#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <stdio.h>
#include "tables.h"

__global__ void test()
{
	uint32_t a = 0xfffefd01, b = 0x00000000, c = 0x00000000;
	b = __byte_perm(a, b, 0x1456);
	// BytePerm(a, 21554);
	printf("------------------------------------\n");
	printf("%08x %08x\n", a, b);
}

void testAES(char* keyBuf)
{
	double kernelSpeed = 0, kernelSpeed2 = 0;
	hipEvent_t start, stop;
	float miliseconds = 0;

	uint32_t* counter, * gpuBuf, * outBuf, *inBuf;
	uint32_t* dev_outBuf, * dev_rk, *dev_inBuf;
	// char* keyBuf;												// Securitz Key
	char* m_EncryptKey = (char*)malloc(16 * 11 * sizeof(char));	// Expanded Keys
 	hipSharedMemConfig pConfig;
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	hipEventCreate(&start);	hipEventCreate(&stop);
	counter = (uint32_t*)malloc(msgSize * sizeof(uint32_t));
	hipHostMalloc((void**)&gpuBuf, msgSize * sizeof(uint32_t));
	hipHostMalloc((void**)&outBuf, msgSize * sizeof(uint32_t));
	hipHostMalloc((void**)&inBuf, msgSize * sizeof(uint32_t));
	// hipHostMalloc((void**)&keyBuf, 16 * sizeof(char));
	hipMalloc((void**)&dev_outBuf, msgSize * sizeof(uint32_t));
	hipMalloc((void**)&dev_inBuf, msgSize * sizeof(uint32_t));
	hipMalloc((void**)&dev_rk, 60 * sizeof(uint32_t));	// AES-128 use 44

	memset(outBuf, msgSize * sizeof(uint32_t), 0);
	memset(counter, msgSize * sizeof(uint32_t), 0);
	hipMemset(dev_outBuf, 0, msgSize * sizeof(uint32_t));

	// //key for test vector, FIPS-197 0x000102030405060708090A0B0C0D0E0F
	// for (int i = 0; i < 16; i++) keyBuf[i] = i;

	if (counter == NULL || gpuBuf == NULL || outBuf == NULL || keyBuf == NULL)
	{
		printf("Memory Allocatation Failed!");
		return;
	}

	for (int i = 0; i < 11 * 16; i++)	m_EncryptKey[i] = 0;

	AESPrepareKey(m_EncryptKey, keyBuf, 128);

// One-T
	// Allocate Tables
	uint8_t *SAES_d;
	uint32_t *t0, *t1, *t2, *t3, *t4, *t4_0, *t4_1, *t4_2, *t4_3, *pret2, *pret3;
	uint32_t *dt0, *dt1, *dt2, *dt3, *dt4, *dt4_0, *dt4_1, *dt4_2, *dt4_3, *dev_pret2, *dev_pret3;
	hipHostMalloc((void**)&t0, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t1, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t2, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t3, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t4, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t4_0, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t4_1, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t4_2, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&t4_3, TABLE_SIZE * sizeof(uint32_t));
	hipHostMalloc((void**)&pret3, pret3Size * sizeof(uint32_t));
	hipHostMalloc((void**)&pret2, pret2Size * sizeof(uint32_t));
	hipMalloc((void**)&dt0, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt1, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt2, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt3, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt4, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt4_0, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt4_1, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt4_2, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dt4_3, TABLE_SIZE * sizeof(uint32_t));
	hipMalloc((void**)&dev_pret3, pret3Size * sizeof(uint32_t));
	hipMalloc((void**)&dev_pret2, pret2Size * sizeof(uint32_t));
	hipMallocManaged(&SAES_d, 256 * sizeof(uint8_t));
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];	t1[i] = T1[i];	
		t2[i] = T2[i];	t3[i] = T3[i];	t4[i] = T4[i];
		t4_0[i] = T4_0[i];		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];		t4_3[i] = T4_3[i];
	}
	for (int i = 0; i < 256; i++) SAES_d[i] = SAES[i]; 
	for (int i = 0; i < msgSize; i++) inBuf[i] = i;

	//plaintext for test vector, FIPS-197 0x00112233445566778899AABBCCDDEEFF
	//key 0x000102030405060708090a0b0c0d0e0f
	//ciphertext shoud be 0x69c4e0d86a7b0430d8cdb78070b4c55a
	// counter[0] = 0x00112233;
	// counter[1] = 0x44556677;
	// counter[2] = 0x8899AABB;
	// counter[3] = 0xCCDDEEFF;
	//Test for correctness
	//AES_128_encrypt(outBuf, (uint32_t *)m_EncryptKey, counter);
	//printf("Input data :   ");
	//printf("%x%x%x%x\n", counter[0], counter[1], counter[2], counter[3]);
	//printf("Output data :   ");
	//printf("%x%x%x%x\n", outBuf[0], outBuf[1], outBuf[2], outBuf[3]);
	//printf("\nMessage size: %d Bytes", msgSize*4);
	// Create an array of counter to encrypt
#ifdef DEBUG
	printf("\n|	Encryption in CPU: Started	|\n");
	
	for (int i = 0; i < msgSize / 4; i++)
	{
		// AES_128_encrypt_CTR(outBuf + 4 * i, (uint32_t*)m_EncryptKey, i, inBuf + 4 * i);
		AES_128_encrypt_CTR(outBuf + i, (uint32_t*)m_EncryptKey, i, inBuf + i);		
	}

	printf("\nOutput data (First 32 Bytes):   \n");
	printf("%x%x%x%x\n", outBuf[0], outBuf[1], outBuf[2], outBuf[3]);	
	printf("%x%x%x%x\n", outBuf[4], outBuf[5], outBuf[6], outBuf[7]);

	printf("\n|	Encryption in GPU: Started	|\n");
#endif
		
	//	 For GPU version, we do not pass in the counter array, because we can use the threadIdx as counter value for free!
		hipMemset(dev_outBuf, 0, msgSize * sizeof(uint32_t));
		for(int i=0; i<msgSize; i++)	gpuBuf[i] = 0;

		 //Coarse grain -  One T-box
		for(int i=0; i<ITERATION; i++)
		{
			hipMemcpy(dev_rk, m_EncryptKey, 60 * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dev_inBuf, inBuf, msgSize*sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dt0, t0, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dt1, t1, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);	
			hipMemcpy(dt2, t2, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dt3, t3, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);						
			hipMemcpy(dt4_0, t4_0, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dt4_1, t4_1, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dt4_2, t4_2, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dt4_3, t4_3, TABLE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
			hipMemcpy(dev_pret2, pret2, pret2Size*sizeof(uint32_t), hipMemcpyHostToDevice);			
			hipMemcpy(dev_pret3, pret3, pret3Size*sizeof(uint32_t), hipMemcpyHostToDevice);				
			
			hipEventRecord(start);
			for (int i = 0; i < msgSize / 4; i+=16777216)
				AES_128_encrypt_CTR_pret3(outBuf + i, (uint32_t*)m_EncryptKey, i, inBuf + i, pret2, pret3);		
			for (int i = 0; i < msgSize / 4; i+=65536)
				AES_128_encrypt_CTR_pret2(outBuf + i, (uint32_t*)m_EncryptKey, i, inBuf + i, pret2, pret3);		
			OneTblBytePermReuseUnroll<<<gridSize/REPEAT, threadSize>>>(dev_outBuf, dev_rk, dt0, dev_inBuf, dev_pret2, dev_pret3); // fastest
			// OneTblBytePermOri<<<gridSize, threadSize>>>(dev_outBuf, dev_rk, dt0, dt4_0, dt4_1, dt4_2, dt4_3, dev_inBuf); // faster
			// OneTblBytePermSBoxOri << <gridSize, threadSize>> > (dev_outBuf, dev_rk, t0, t4, SAES_d, dev_inBuf);		// slow
			// OneTblBytePermSBoxComb << <gridSize, threadSize>> > (dev_outBuf, dev_rk, t0, t4, SAES_d, dev_inBuf);	// slow			
			hipEventSynchronize(stop);
			hipEventRecord(stop);
			hipMemcpy(gpuBuf, dev_outBuf, msgSize*sizeof(uint32_t), hipMemcpyDeviceToHost);
			hipEventElapsedTime(&miliseconds, start, stop);
			kernelSpeed2 += 8*(4*(msgSize/1024)) / (miliseconds);
		}
		printf("\nAES GPU (one-T): %u MB of data. Kernel: %.4f [Gbps]\n", 4*(msgSize/1024/1024), kernelSpeed2/1024/ITERATION);
		printf("GPU (one-T) Output data (First 32 Bytes):   \n");
		printf("%x%x%x%x\n", gpuBuf[0], gpuBuf[1], gpuBuf[2], gpuBuf[3]);
		printf("%x%x%x%x\n", gpuBuf[4], gpuBuf[5], gpuBuf[6], gpuBuf[7]);

#ifdef DEBUG
	for (int i = 0; i < msgSize; i++)
	{
		if (gpuBuf[i] != outBuf[i])
		{
			printf("AES wrong at %d gpu: %x cpu: %x\n", i, gpuBuf[i], outBuf[i]);
			return;
		}
	}
	printf("The results in CPU and GPU match!\n");
#endif
}

int main(int argc, char** argv)
{
	int i, j;
	char* user_key = (char*) malloc(16*sizeof(char));
	printf("<------ TESTING AES-128 CTR Mode ------>\n");

	if(argc==1)
	{
		printf("Use Default Key:\n");
		//key for test vector, FIPS-197 0x000102030405060708090A0B0C0D0E0F
		for(j=0; j<16; j++) user_key[j] = j;
	}
	else if(argc==2)
	{
		printf("New User Key:\n");
		strcpy(user_key, argv[1]);
  		for(j=0; j<16; j++) printf("%c ", user_key[j]);
  	}
  	else
  	{
  		printf("Wrong Arguments!\n");
  		return 0;
  	}


	hipSharedMemConfig pConfig;
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);// Avoid bank conflict for 64 bit access. 
	hipDeviceGetSharedMemConfig(&pConfig);
	//printf("Share mem config: %d\n", pConfig);
	hipDeviceSetCacheConfig(hipFuncCachePreferNone);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("\nGPU Compute Capability = [%d.%d], clock: %d asynCopy: %d MapHost: %d SM: %d\n",
		deviceProp.major, deviceProp.minor, deviceProp.clockRate, deviceProp.asyncEngineCount, deviceProp.canMapHostMemory, deviceProp.multiProcessorCount);
	printf("msgSize: %lu MB\t counter blocks: %u M Block\n", msgSize * 4 / 1024 / 1024, msgSize / 1024 / 1024);
	printf("%u blocks and %u threads\n", gridSize, threadSize);
	testAES(user_key);
	// hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();


	return 0;

}
